#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include "ndCudaStdafx.h"
#include "ndCudaDevice.h"
#include "ndCudaContext.h"
#include "ndCudaSphFluid.h"

__global__ void ndFluidInitTranspose(const ndKernelParams params, const ndAssessor<ndCudaVector> input, ndSphFliudPoint::ndPointAssessor output)
{
	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int blockSride = blockDim.x;

	int base = blockSride * params.m_blocksPerKernel * blockId;
	for (int i = 0; i < params.m_blocksPerKernel; ++i)
	{
		int index = base + threadId;
		if (index < input.m_size)
		{
			ndCudaVector point(input[index]);
			output.m_x[index] = point.x;
			output.m_y[index] = point.y;
			output.m_z[index] = point.z;
		}

		base += blockSride;
	}
};

__global__ void ndFluidGetPositions(const ndKernelParams params, ndAssessor<ndCudaVector> output, const ndSphFliudPoint::ndPointAssessor input)
{
	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int blockSride = blockDim.x;

	int base = blockSride * params.m_blocksPerKernel * blockId;
	for (int i = 0; i < params.m_blocksPerKernel; ++i)
	{
		int index = base + threadId;
		if (index < output.m_size)
		{
			ndCudaVector point(input.m_x[index], input.m_y[index], input.m_z[index], 1.0f);
			output[index] = point;
		}
		base += blockSride;
	}
};


__global__ void ndCalculateAabb(const ndSphFliudPoint::ndPointAssessor input)
{
//	D_TRACKTIME_NAMED(CalculateAabb);
//	ndBox box;
//	const ndArray<ndVector>& posit = m_posit;
//	const ndStartEnd startEnd(posit.GetCount(), threadIndex, threadCount);
//	for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
//	{
//		box.m_min = box.m_min.GetMin(posit[i]);
//		box.m_max = box.m_max.GetMax(posit[i]);
//	}
//	boxes[threadIndex] = box;

	int threadId = threadIdx.x;
	int blockSride = blockDim.x;
	int blockId = blockIdx.x;
	int index = threadId + blockSride * blockId;
	if (index < input.m_x.m_size)
	{
		ndCudaVector point(input.m_x[threadId], input.m_y[threadId], input.m_z[threadId], 1.0f);
	}

}

ndCudaSphFliud::ndCudaSphFliud(ndCudaContext* const context, ndBodySphFluid* const owner)
	:m_owner(owner)
	,m_context(context)
	,m_points()
{
}

ndCudaSphFliud::~ndCudaSphFliud()
{
}

void ndCudaSphFliud::MemCpy(const double* const src, int strideInItems, int items)
{
	ndAssert(0);
}

void GetPositions(double* const dst, int strideInItems, int items)
{
	ndAssert(0);
}

void ndCudaSphFliud::MemCpy(const float* const src, int strideInItems, int items)
{
	m_points.SetCount(items);

	if (strideInItems == sizeof(ndCudaVector) / sizeof(float))
	{
		const ndCudaVector* const srcPtr = (ndCudaVector*)src;
		m_points.ReadData(srcPtr, items);
	}
	else
	{
		ndAssert(0);
	}

	InitBuffers();
}

void ndCudaSphFliud::GetPositions(float* const dst, int strideInItems, int items)
{
	ndAssessor<ndCudaVector> output(m_points);
	const ndSphFliudPoint::ndPointAssessor input(m_workingPoint);
	const ndKernelParams params(m_context->m_device, m_context->m_device->m_workGroupSize, m_points.GetCount());

	if (strideInItems == sizeof(ndCudaVector) / sizeof(float))
	{
		ndFluidGetPositions << <params.m_kernelCount, params.m_workGroupSize, 0 >> > (params, output, input);
		ndCudaVector* const dstPtr = (ndCudaVector*)dst;
		m_points.WriteData(dstPtr, items);
	}
	else
	{
		ndAssert(0);
	}
}

void ndCudaSphFliud::InitBuffers()
{
	const ndKernelParams params(m_context->m_device, m_context->m_device->m_workGroupSize, m_points.GetCount());

	//m_aabb.SetCount(groups);
	//m_aabb.SetCount(params.m_kernelCount + 32);
	m_workingPoint.m_x.SetCount(params.m_itemCount);
	m_workingPoint.m_y.SetCount(params.m_itemCount);
	m_workingPoint.m_z.SetCount(params.m_itemCount);

	const ndAssessor<ndCudaVector> input(m_points);
	ndSphFliudPoint::ndPointAssessor output(m_workingPoint);
	ndFluidInitTranspose<<<params.m_kernelCount, params.m_workGroupSize, 0>>>(params, input, output);
}

void ndCudaSphFliud::Update(float timestep)
{
	CaculateAabb();
}

void ndCudaSphFliud::CaculateAabb()
{
	//D_TRACKTIME();
	//class ndBox
	//{
	//	public:
	//	ndBox()
	//		:m_min(ndFloat32(1.0e10f))
	//		, m_max(ndFloat32(-1.0e10f))
	//	{
	//	}
	//	ndVector m_min;
	//	ndVector m_max;
	//};
	//
	//ndBox boxes[D_MAX_THREADS_COUNT];
	//auto CalculateAabb = ndMakeObject::ndFunction([this, &boxes](ndInt32 threadIndex, ndInt32 threadCount)
	//{
	//	D_TRACKTIME_NAMED(CalculateAabb);
	//	ndBox box;
	//	const ndArray<ndVector>& posit = m_posit;
	//	const ndStartEnd startEnd(posit.GetCount(), threadIndex, threadCount);
	//	for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
	//	{
	//		box.m_min = box.m_min.GetMin(posit[i]);
	//		box.m_max = box.m_max.GetMax(posit[i]);
	//	}
	//	boxes[threadIndex] = box;
	//});
	//
	//threadPool->ParallelExecute(CalculateAabb);
	//
	//ndBox box;
	//const ndInt32 threadCount = threadPool->GetThreadCount();
	//for (ndInt32 i = 0; i < threadCount; ++i)
	//{
	//	box.m_min = box.m_min.GetMin(boxes[i].m_min);
	//	box.m_max = box.m_max.GetMax(boxes[i].m_max);
	//}
	//
	//const ndFloat32 gridSize = GetSphGridSize();
	//
	//ndVector grid(gridSize);
	//ndVector invGrid(ndFloat32(1.0f) / gridSize);
	//
	//// add one grid padding to the aabb
	//box.m_min -= grid;
	//box.m_max += (grid + grid);
	//
	//// quantize the aabb to integers of the gird size
	//box.m_min = grid * (box.m_min * invGrid).Floor();
	//box.m_max = grid * (box.m_max * invGrid).Floor();
	//
	//// make sure the w component is zero.
	//m_box0 = box.m_min & ndVector::m_triplexMask;
	//m_box1 = box.m_max & ndVector::m_triplexMask;
	//
	//ndWorkingBuffers& data = *m_workingBuffers;
	//ndInt32 numberOfGrid = ndInt32((box.m_max.m_x - box.m_min.m_x) * invGrid.m_x + ndFloat32(1.0f));
	//data.SetWorldToGridMapping(numberOfGrid, m_box1.m_x, m_box0.m_x);

	ndSphFliudPoint::ndPointAssessor output(m_workingPoint);
}

