#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include "ndCudaStdafx.h"
#include "ndCudaUtils.h"
#include "ndCudaContext.h"
#include "ndCudaPrefixScan.cuh"
#include "ndCudaCountingSort.cuh"
#include "ndCudaContextImplement.h"

#define D_CUDA_SCENE_GRID_SIZE		8.0f
#define D_CUDA_SCENE_INV_GRID_SIZE	(1.0f/D_CUDA_SCENE_GRID_SIZE) 

__global__ void ndCudaBeginFrame(ndCudaSceneInfo& info)
{
	long long coreTicks = clock64();
	info.m_timeSlice = coreTicks;
	//printf("t0 = %lld    ", coreTicks);
}

__global__ void ndCudaEndFrame(ndCudaSceneInfo& info, int frameCount)
{
	long long coreTicks = clock64();

	info.m_frameCount = frameCount;
	long long diff = (coreTicks >= info.m_timeSlice) ? (coreTicks - info.m_timeSlice) : (info.m_timeSlice - coreTicks);
	//printf("t1 = %lld   diff= %lld\n", coreTicks, coreTicks - info.m_timeSlice);
	//printf("t1 = %lld   diff= %lld\n", coreTicks, x);

	//info.m_timeSlice = coreTicks - info.m_timeSlice;
	info.m_timeSlice = diff;
}

__global__ void ndCudaInitTransforms(ndCudaSceneInfo& info)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index < info.m_bodyArray.m_size)
	{
		const ndCudaBodyProxy* src = info.m_bodyArray.m_array;
		ndCudaSpatialVector* dst0 = info.m_transformBuffer0.m_array;
		ndCudaSpatialVector* dst1 = info.m_transformBuffer1.m_array;
	
		dst0[index].m_linear = src[index].m_posit;
		dst0[index].m_angular = src[index].m_rotation;
		dst1[index].m_linear = src[index].m_posit;
		dst1[index].m_angular = src[index].m_rotation;
	}
}

__global__ void ndCudaInitBodyArray(ndCudaSceneInfo& info)
{
	__shared__  ndCudaBoundingBox cacheAabb[D_THREADS_PER_BLOCK];

	const unsigned threadId = threadIdx.x;
	const unsigned index = threadId + blockDim.x * blockIdx.x;
	const unsigned bodyCount = info.m_bodyArray.m_size - 1;

	//if (index == 0)
	//{
	//	printf("InitBodyArray\n");
	//}

	if (index < bodyCount)
	{
		ndCudaBodyProxy* bodyArray = info.m_bodyArray.m_array;
		ndCudaBodyProxy& body = bodyArray[index];

		// calculate shape global Matrix
		body.m_globalSphapeRotation = body.m_localRotation * body.m_rotation;
		ndCudaMatrix3x3 matrix(body.m_globalSphapeRotation.GetMatrix3x3());
		body.m_globalSphapePosition = matrix.RotateVector(body.m_localPosition) + body.m_posit;

		matrix.m_front = matrix.m_front.Scale(body.m_scale.x);
		matrix.m_up = matrix.m_up.Scale(body.m_scale.y);
		matrix.m_right = matrix.m_right.Scale(body.m_scale.z);
		matrix = body.m_alignRotation.GetMatrix3x3() * matrix;

		const ndCudaVector origin(matrix.RotateVector(body.m_obbOrigin) + body.m_globalSphapePosition);
		const ndCudaVector size(matrix.m_front.Abs().Scale(body.m_obbSize.x) + matrix.m_up.Abs().Scale(body.m_obbSize.y) + matrix.m_right.Abs().Scale(body.m_obbSize.z));

		const ndCudaVector padding(1.0f / 16.0f);
		const ndCudaVector minBox(origin - size - padding);
		const ndCudaVector maxBox(origin + size + padding);

		// save aabb and calculate bonding box for this thread block
		body.m_minAabb = minBox;
		body.m_maxAabb = maxBox;
		cacheAabb[threadId].m_min = minBox;
		cacheAabb[threadId].m_max = maxBox;
	}

	const unsigned lastBlock = bodyCount / D_THREADS_PER_BLOCK;
	if (lastBlock == blockIdx.x)
	{
		__syncthreads();
		const unsigned lastId = bodyCount - D_THREADS_PER_BLOCK * lastBlock;
		const ndCudaBoundingBox box(cacheAabb[0]);
		if (threadId >= lastId)
		{
			cacheAabb[threadId] = box;
		}
	}
	__syncthreads();

	ndCudaBoundingBox* bBox = info.m_bodyAabbArray.m_array;
	for (int i = D_THREADS_PER_BLOCK / 2; i; i = i >> 1)
	{
		if (threadId < i)
		{
			cacheAabb[threadId].m_min = cacheAabb[threadId].m_min.Min(cacheAabb[threadId + i].m_min);
			cacheAabb[threadId].m_max = cacheAabb[threadId].m_max.Max(cacheAabb[threadId + i].m_max);
		}
		__syncthreads();
	}

	if (threadId == 0)
	{
		bBox[blockIdx.x].m_min = cacheAabb[0].m_min;
		bBox[blockIdx.x].m_max = cacheAabb[0].m_max;
	}
};

__global__ void ndCudaMergeAabb(ndCudaSceneInfo& info)
{
	__shared__  ndCudaBoundingBox cacheAabb[D_THREADS_PER_BLOCK];

	const ndCudaBoundingBox* bBoxOut = info.m_bodyAabbArray.m_array;

	const unsigned threadId = threadIdx.x;
	const unsigned boxCount = info.m_bodyAabbArray.m_size - 1;
	const unsigned aabbBlocks = boxCount / D_THREADS_PER_BLOCK;
	const unsigned boxLastRow = boxCount - aabbBlocks * D_THREADS_PER_BLOCK;

	//if (threadId == 0)
	//{
	//	printf("CudaMergeAabb\n");
	//}

	cacheAabb[threadId] = bBoxOut[0];
	if (threadId < boxLastRow)
	{
		cacheAabb[threadId] = bBoxOut[aabbBlocks * D_THREADS_PER_BLOCK + threadId];
	}
	__syncthreads();

	unsigned base = 0;
	for (int i = 0; i < aabbBlocks; i++)
	{
		cacheAabb[threadId].m_min = cacheAabb[threadId].m_min.Min(cacheAabb[base + threadId].m_min);
		cacheAabb[threadId].m_max = cacheAabb[threadId].m_max.Min(cacheAabb[base + threadId].m_max);
		base += D_THREADS_PER_BLOCK;
	}

	__syncthreads();
	for (int i = D_THREADS_PER_BLOCK / 2; i; i = i >> 1)
	{
		if (threadId < i)
		{
			cacheAabb[threadId].m_min = cacheAabb[threadId].m_min.Min(cacheAabb[threadId + i].m_min);
			cacheAabb[threadId].m_max = cacheAabb[threadId].m_max.Max(cacheAabb[threadId + i].m_max);
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		ndCudaVector minBox((cacheAabb[0].m_min.Scale(D_CUDA_SCENE_INV_GRID_SIZE).Floor()).Scale(D_CUDA_SCENE_GRID_SIZE));
		ndCudaVector maxBox((cacheAabb[0].m_max.Scale(D_CUDA_SCENE_INV_GRID_SIZE).Floor()).Scale(D_CUDA_SCENE_GRID_SIZE) + ndCudaVector(D_CUDA_SCENE_GRID_SIZE));
		minBox.w = 0.0f;
		maxBox.w = 0.0f;
		info.m_worldBox.m_min = minBox;
		info.m_worldBox.m_max = maxBox;
	}
};

__global__ void ndCudaCountAabb(ndCudaSceneInfo& info)
{
	__shared__  unsigned cacheBuffer[D_THREADS_PER_BLOCK / 2 + D_THREADS_PER_BLOCK];

	const unsigned blockId = blockIdx.x;
	const unsigned bodyCount = info.m_bodyArray.m_size - 1;
	const unsigned blocks = (bodyCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	if (blockId < blocks)
	{
		const unsigned threadId = threadIdx.x;
		const unsigned threadId1 = D_THREADS_PER_BLOCK / 2 + threadId;
		const unsigned index = threadId + blockDim.x * blockId;

		cacheBuffer[threadId] = 0;
		cacheBuffer[threadId1] = 0;
		if (index < bodyCount)
		{
			ndCudaBodyProxy* bodyArray = info.m_bodyArray.m_array;

			const ndCudaVector minBox(info.m_worldBox.m_min);
			const ndCudaVector bodyBoxMin(bodyArray[index].m_minAabb);
			const ndCudaVector bodyBoxMax(bodyArray[index].m_maxAabb);

			const int x0 = __float2int_rd((bodyBoxMin.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int y0 = __float2int_rd((bodyBoxMin.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int z0 = __float2int_rd((bodyBoxMin.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int x1 = __float2int_rd((bodyBoxMax.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
			const int y1 = __float2int_rd((bodyBoxMax.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
			const int z1 = __float2int_rd((bodyBoxMax.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
			const int count = (z1 - z0) * (y1 - y0) * (x1 - x0);
			cacheBuffer[threadId1] = count;
		}
		__syncthreads();

		for (int i = 1; i < D_THREADS_PER_BLOCK; i = i << 1)
		{
			int sum = cacheBuffer[threadId1] + cacheBuffer[threadId1 - i];
			__syncthreads();
			cacheBuffer[threadId1] = sum;
			__syncthreads();
		}

		const unsigned newCapacity = D_PREFIX_SCAN_PASSES * D_THREADS_PER_BLOCK * ((blocks + D_PREFIX_SCAN_PASSES - 1) / D_PREFIX_SCAN_PASSES) + D_THREADS_PER_BLOCK;
		if (newCapacity >= info.m_histogram.m_capacity)
		{
			if (index == 0)
			{
				#ifdef _DEBUG
				printf("function: CudaCountAabb: histogram buffer overflow\n");
				#endif
			}
			info.m_frameIsValid = 0;
			info.m_histogram.m_size = info.m_histogram.m_capacity + 1;
		}
		else
		{
			unsigned* histogram = info.m_histogram.m_array;
			histogram[index] = cacheBuffer[threadId1];
			if (index == 0)
			{
				info.m_histogram.m_size = blocks * D_THREADS_PER_BLOCK;
			}
		}
	}
};

__global__ void ndCudaGenerateHashGrids(const ndCudaSceneInfo& info)
{
	const unsigned threadId = threadIdx.x;
	const unsigned index = threadId + blockDim.x * blockIdx.x;
	const unsigned bodyCount = info.m_bodyArray.m_size - 1;
	if (index < bodyCount)
	{
		const unsigned* histogram = info.m_histogram.m_array;
		const ndCudaBodyProxy* bodyArray = info.m_bodyArray.m_array;
		ndCudaBodyAabbCell* hashArray = info.m_bodyAabbCell.m_array;

		const ndCudaVector minBox(info.m_worldBox.m_min);
		const ndCudaVector bodyBoxMin(bodyArray[index].m_minAabb);
		const ndCudaVector bodyBoxMax(bodyArray[index].m_maxAabb);

		const int x0 = __float2int_rd((bodyBoxMin.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE);
		const int y0 = __float2int_rd((bodyBoxMin.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE);
		const int z0 = __float2int_rd((bodyBoxMin.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE);
		const int x1 = __float2int_rd((bodyBoxMax.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
		const int y1 = __float2int_rd((bodyBoxMax.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
		const int z1 = __float2int_rd((bodyBoxMax.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;

		ndCudaBodyAabbCell hash;
		hash.m_id = index;
		hash.m_key = 0;
		unsigned start = index ? histogram[index - 1] : 0;

		for (int z = z0; z < z1; z++)
		{
			hash.m_z = z;
			for (int y = y0; y < y1; y++)
			{
				hash.m_y = y;
				for (int x = x0; x < x1; x++)
				{
					hash.m_x = x;
					hashArray[start] = hash;
					start++;
				}
			}
		}
	}
};

__global__ void ndCudaScene(ndCudaSceneInfo& info)
{
	unsigned bodyCount = info.m_bodyArray.m_size - 1;
	unsigned bodyBlocksCount = (bodyCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;

	//printf("ndCudaScene %d\n", bodyBlocksCount);
	ndCudaInitBodyArray << <bodyBlocksCount, D_THREADS_PER_BLOCK, 0 >> > (info);
	ndCudaMergeAabb << <1, D_THREADS_PER_BLOCK, 0 >> > (info);
	ndCudaCountAabb << <bodyBlocksCount, D_THREADS_PER_BLOCK, 0 >> > (info);
	if (info.m_frameIsValid == 0)
	{
		return;
	}

	//printf("ndCudaScene %d\n", bodyBlocksCount);
	ndCudaHillisSteelePrefixScan << <1, 1, 0 >> > (info, D_THREADS_PER_BLOCK);

	//const unsigned lastIndex = info.m_bodyArray.m_size - 2;
	const unsigned* histogram = info.m_histogram.m_array;
	const unsigned cellCount = histogram[bodyCount - 1];
	if ((cellCount + D_THREADS_PER_BLOCK) > info.m_bodyAabbCellScrath.m_capacity)
	{
		#ifdef _DEBUG
		printf("function: ValidateGridArray: histogram buffer overflow\n");
		#endif
		info.m_frameIsValid = 0;
		info.m_bodyAabbCell.m_size = cellCount + D_THREADS_PER_BLOCK;
		info.m_bodyAabbCellScrath.m_size = cellCount + D_THREADS_PER_BLOCK;
		return;
	}

	ndCudaBodyAabbCell* hashArray = info.m_bodyAabbCell.m_array;
	ndCudaBodyAabbCell* hashArrayScrath = info.m_bodyAabbCellScrath.m_array;
	ndCudaBodyAabbCell hash;
	hash.m_value = 0;
	hash.m_id = unsigned(-1);
	hash.m_x = unsigned(-1);
	hash.m_y = unsigned(-1);
	hash.m_z = unsigned(-1);
	const long long value = hash.m_value;
	hashArray[cellCount].m_value = value;
	hashArrayScrath[cellCount].m_value = value;

	info.m_bodyAabbCell.m_size = cellCount + 1;
	info.m_bodyAabbCellScrath.m_size = cellCount + 1;

	// check new histogram size.
	const unsigned newCapacity = ndCudaCountingSortCalculateScanPrefixSize(cellCount, D_THREADS_PER_BLOCK);
	if (newCapacity >= info.m_histogram.m_capacity)
	{
		#ifdef _DEBUG
		printf("function: ValidateGridArray: histogram buffer overflow\n");
		#endif
		info.m_frameIsValid = 0;
		info.m_histogram.m_size = newCapacity;
	}
	//const unsigned blocksCount = (cellCount + D_COUNTING_SORT_MAX_BLOCK_SIZE - 1) / D_COUNTING_SORT_MAX_BLOCK_SIZE;
	info.m_histogram.m_size = bodyBlocksCount * D_THREADS_PER_BLOCK;

	//printf("ndCudaScene %d  %d\n", info.m_histogram.m_size, info.m_histogram.m_capacity);
	ndCudaGenerateHashGrids << <bodyBlocksCount, D_THREADS_PER_BLOCK, 0 >> > (info);
}

template <typename SortKeyPredicate>
__global__ void ndCudaSortGridArray(ndCudaSceneInfo& info, SortKeyPredicate sortKey_x, SortKeyPredicate sortKey_y, SortKeyPredicate sortKey_z, SortKeyPredicate sortKey_w)
{
	if (info.m_frameIsValid)
	{
		const unsigned size = info.m_bodyAabbCell.m_size - 1;
		unsigned* histogram = info.m_histogram.m_array;

		printf("%d %d %d %d\n", histogram[0], histogram[1], histogram[2], histogram[3]);

		ndCudaBodyAabbCell* src = info.m_bodyAabbCell.m_array;
		ndCudaBodyAabbCell* dst = info.m_bodyAabbCellScrath.m_array;
		ndCudaCountingSort << <1, 1, 0 >> > (info, src, dst, histogram, size, sortKey_x, D_THREADS_PER_BLOCK);
		//ndCudaCountingSort << <1, 1, 0 >> > (info, dst, src, histogram, size, sortKey_y, D_THREADS_PER_BLOCK);
		//ndCudaCountingSort << <1, 1, 0 >> > (info, src, dst, histogram, size, sortKey_z, D_THREADS_PER_BLOCK);
		//ndCudaCountingSort << <1, 1, 0 >> > (info, dst, src, histogram, size, sortKey_w, D_THREADS_PER_BLOCK);
	}
}

ndCudaContextImplement::ndCudaContextImplement()
	:m_sceneInfoGpu(nullptr)
	,m_sceneInfoCpu(nullptr)
	,m_histogram()
	,m_bodyBufferGpu()
	,m_bodyAabbCell()
	,m_bodyAabbCellScrath()
	,m_boundingBoxGpu()
	,m_transformBufferCpu0()
	,m_transformBufferCpu1()
	,m_transformBufferGpu0()
	,m_transformBufferGpu1()
	,m_solverMemCpyStream(0)
	,m_solverComputeStream(0)
	,m_frameCounter(0)
{
	hipError_t cudaStatus;
	cudaStatus = hipStreamCreate(&m_solverMemCpyStream);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipStreamCreate(&m_solverComputeStream);
	dAssert(cudaStatus == hipSuccess);
	
	cudaStatus = hipMalloc((void**)&m_sceneInfoGpu, sizeof(ndCudaSceneInfo));
	dAssert(cudaStatus == hipSuccess);
	
	cudaStatus = hipHostMalloc((void**)&m_sceneInfoCpu, sizeof(ndCudaSceneInfo));
	dAssert(cudaStatus == hipSuccess);
	
	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
	
	*m_sceneInfoCpu = ndCudaSceneInfo();
}

ndCudaContextImplement::~ndCudaContextImplement()
{
	hipError_t cudaStatus;
	
	cudaStatus = hipHostFree(m_sceneInfoCpu);
	dAssert(cudaStatus == hipSuccess);
	
	cudaStatus = hipFree(m_sceneInfoGpu);
	dAssert(cudaStatus == hipSuccess);
	
	cudaStatus = hipStreamDestroy(m_solverComputeStream);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipStreamDestroy(m_solverMemCpyStream);
	dAssert(cudaStatus == hipSuccess);
	
	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

long long ndCudaContextImplement::GetGpuClocks() const
{
	return m_sceneInfoCpu->m_timeSlice;
}

void ndCudaContextImplement::SwapBuffers()
{
	m_frameCounter = m_frameCounter + 1;

	ndCudaSceneInfo* const gpuInfo = m_sceneInfoGpu;
	ndCudaEndFrame << < 1, 1, 0, m_solverComputeStream >> > (*gpuInfo, m_frameCounter);
	m_transformBufferCpu0.Swap(m_transformBufferCpu1);
}

void ndCudaContextImplement::Begin()
{
	hipDeviceSynchronize();

	// get the scene info from the update	
	ndCudaSceneInfo* const gpuInfo = m_sceneInfoGpu;
	ndCudaSceneInfo* const cpuInfo = m_sceneInfoCpu;

	hipError_t cudaStatus = hipMemcpyAsync(cpuInfo, gpuInfo, sizeof(ndCudaSceneInfo), hipMemcpyDeviceToHost, m_solverMemCpyStream);
	dAssert(cudaStatus == hipSuccess);
	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}

	const int frameCounter = m_frameCounter;
	//ndCudaEndFrame << < 1, 1, 0, m_solverComputeStream >> > (*gpuInfo, frameCounter);
	if (frameCounter)
	{
		ndCudaHostBuffer<ndCudaSpatialVector>& cpuBuffer = m_transformBufferCpu0;
		ndCudaDeviceBuffer<ndCudaSpatialVector>& gpuBuffer = (frameCounter & 1) ? m_transformBufferGpu1 : m_transformBufferGpu0;
		gpuBuffer.WriteData(&cpuBuffer[0], cpuBuffer.GetCount() - 1, m_solverMemCpyStream);
	}

	ndCudaBeginFrame << <1, 1, 0, m_solverComputeStream >> > (*gpuInfo);
}

ndCudaSpatialVector* ndCudaContextImplement::GetTransformBuffer0()
{
	return &m_transformBufferCpu0[0];
}

ndCudaSpatialVector* ndCudaContextImplement::GetTransformBuffer1()
{
	return &m_transformBufferCpu1[0];
}

void ndCudaContextImplement::ResizeBuffers(int cpuBodyCount)
{
	const int gpuBodyCount = D_THREADS_PER_BLOCK * ((cpuBodyCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK);
	
	ndCudaDeviceBuffer<unsigned>& histogramGpu = m_histogram;
	ndCudaDeviceBuffer<ndCudaBodyProxy>& bodyBufferGpu = m_bodyBufferGpu;
	ndCudaDeviceBuffer<ndCudaBoundingBox>& boundingBoxGpu = m_boundingBoxGpu;
	ndCudaDeviceBuffer<ndCudaBodyAabbCell>& bodyAabbCellGpu0 = m_bodyAabbCell;
	ndCudaDeviceBuffer<ndCudaBodyAabbCell>& bodyAabbCellGpu1 = m_bodyAabbCellScrath;
	ndCudaHostBuffer<ndCudaSpatialVector>& transformBufferCpu0 = m_transformBufferCpu0;
	ndCudaHostBuffer<ndCudaSpatialVector>& transformBufferCpu1 = m_transformBufferCpu1;
	ndCudaDeviceBuffer<ndCudaSpatialVector>& transformBufferGpu0 = m_transformBufferGpu0;
	ndCudaDeviceBuffer<ndCudaSpatialVector>& transformBufferGpu1 = m_transformBufferGpu1;
	
	histogramGpu.SetCount(cpuBodyCount);
	bodyBufferGpu.SetCount(cpuBodyCount);
	bodyAabbCellGpu0.SetCount(cpuBodyCount);
	bodyAabbCellGpu1.SetCount(cpuBodyCount);
	transformBufferGpu0.SetCount(cpuBodyCount);
	transformBufferGpu1.SetCount(cpuBodyCount);
	transformBufferCpu0.SetCount(cpuBodyCount);
	transformBufferCpu1.SetCount(cpuBodyCount);
	boundingBoxGpu.SetCount(gpuBodyCount / D_THREADS_PER_BLOCK);
}

void ndCudaContextImplement::LoadBodyData(const ndCudaBodyProxy* const src, int cpuBodyCount)
{
	hipDeviceSynchronize();
		
	ndCudaSceneInfo info;
	info.m_histogram = ndCudaBuffer<unsigned>(m_histogram);
	info.m_bodyArray = ndCudaBuffer<ndCudaBodyProxy>(m_bodyBufferGpu);
	info.m_bodyAabbArray = ndCudaBuffer<ndCudaBoundingBox>(m_boundingBoxGpu);
	info.m_bodyAabbCell = ndCudaBuffer<ndCudaBodyAabbCell>(m_bodyAabbCell);
	info.m_bodyAabbCellScrath = ndCudaBuffer<ndCudaBodyAabbCell>(m_bodyAabbCellScrath);
	info.m_transformBuffer0 = ndCudaBuffer<ndCudaSpatialVector>(m_transformBufferGpu0);
	info.m_transformBuffer1 = ndCudaBuffer<ndCudaSpatialVector>(m_transformBufferGpu0);
	
	*m_sceneInfoCpu = info;
	hipError_t cudaStatus = hipMemcpy(m_sceneInfoGpu, &info, sizeof(ndCudaSceneInfo), hipMemcpyHostToDevice);
	dAssert(cudaStatus == hipSuccess);

	const int blocksCount = (cpuBodyCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	//const int gpuBodyCount = D_THREADS_PER_BLOCK * ((cpuBodyCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK);
	
	m_bodyBufferGpu.ReadData(src, cpuBodyCount);
	ndCudaInitTransforms << <blocksCount, D_THREADS_PER_BLOCK, 0, 0 >> > (*m_sceneInfoCpu);
	
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

void ndCudaContextImplement::ValidateContextBuffers()
{
	ndCudaSceneInfo* const sceneInfo = m_sceneInfoCpu;
	if (!sceneInfo->m_frameIsValid)
	{
		hipDeviceSynchronize();
		sceneInfo->m_frameIsValid = 1;

		if (sceneInfo->m_histogram.m_size > sceneInfo->m_histogram.m_capacity)
		{
			m_histogram.SetCount(sceneInfo->m_histogram.m_size);
			sceneInfo->m_histogram = ndCudaBuffer<unsigned>(m_histogram);
		}

		if (sceneInfo->m_bodyAabbCell.m_size > sceneInfo->m_bodyAabbCell.m_capacity)
		{
			m_bodyAabbCell.SetCount(sceneInfo->m_bodyAabbCell.m_size);
			m_bodyAabbCellScrath.SetCount(sceneInfo->m_bodyAabbCell.m_size);
			sceneInfo->m_bodyAabbCell = ndCudaBuffer<ndCudaBodyAabbCell>(m_bodyAabbCell);
			sceneInfo->m_bodyAabbCellScrath = ndCudaBuffer<ndCudaBodyAabbCell>(m_bodyAabbCellScrath);
		}

		hipError_t cudaStatus = hipMemcpy(m_sceneInfoGpu, sceneInfo, sizeof(ndCudaSceneInfo), hipMemcpyHostToDevice);
		dAssert(cudaStatus == hipSuccess);
		if (cudaStatus != hipSuccess)
		{
			dAssert(0);
		}
		hipDeviceSynchronize();
	}
}

void ndCudaContextImplement::InitBodyArray()
{
	//auto CompactMovingBodies = ndMakeObject::ndFunction([this, &scans](int threadIndex, int threadCount)
	//{
	//	const ndArray<ndBodyKinematic*>& activeBodyArray = GetActiveBodyArray();
	//	ndBodyKinematic** const sceneBodyArray = &m_sceneBodyArray[0];
	//
	//	const ndArray<ndBodyKinematic*>& view = m_bodyList.m_view;
	//	int* const scan = &scans[threadIndex][0];
	//
	//	const ndStartEnd startEnd(view.GetCount(), threadIndex, threadCount);
	//	for (int i = startEnd.m_start; i < startEnd.m_end; ++i)
	//	{
	//		ndBodyKinematic* const body = activeBodyArray[i];
	//		const int key = body->m_sceneEquilibrium;
	//		const int index = scan[key];
	//		sceneBodyArray[index] = body;
	//		scan[key] ++;
	//	}
	//});

	//ParallelExecute(BuildBodyArray);
	//int sum = 0;
	//int threadCount = GetThreadCount();
	//for (int j = 0; j < 2; j++)
	//{
	//	for (int i = 0; i < threadCount; ++i)
	//	{
	//		const int count = scans[i][j];
	//		scans[i][j] = sum;
	//		sum += count;
	//	}
	//}
	//
	//int movingBodyCount = scans[0][1] - scans[0][0];
	//m_sceneBodyArray.SetCount(m_bodyList.GetCount());
	//if (movingBodyCount)
	//{
	//	ParallelExecute(CompactMovingBodies);
	//}
	//
	//m_sceneBodyArray.SetCount(movingBodyCount);
	//
	//ndBodyKinematic* const sentinelBody = m_sentinelBody;
	//sentinelBody->PrepareStep(GetActiveBodyArray().GetCount() - 1);
	//
	//sentinelBody->m_isStatic = 1;
	//sentinelBody->m_autoSleep = 1;
	//sentinelBody->m_equilibrium = 1;
	//sentinelBody->m_equilibrium0 = 1;
	//sentinelBody->m_isJointFence0 = 1;
	//sentinelBody->m_isJointFence1 = 1;
	//sentinelBody->m_isConstrained = 0;
	//sentinelBody->m_sceneEquilibrium = 1;
	//sentinelBody->m_weigh = ndFloat32(0.0f);

#if 0


	auto CalculateBodyPairsCount = [] __device__(cuSceneInfo & info)
	{
		__shared__  unsigned cacheBuffer[D_THREADS_PER_BLOCK / 2 + D_THREADS_PER_BLOCK];

		const unsigned blockId = blockIdx.x;
		const unsigned cellCount = info.m_bodyAabbCell.m_size - 1;
		const unsigned blocks = (cellCount + blockDim.x - 1) / blockDim.x;
		if (blockId < blocks)
		{
			const unsigned threadId = threadIdx.x;
			const unsigned threadId1 = D_THREADS_PER_BLOCK / 2 + threadId;
			int index = threadId + blockDim.x * blockIdx.x;

			cacheBuffer[threadId] = 0;
			cacheBuffer[threadId1] = 0;
			if (index < cellCount)
			{
				const cuBodyAabbCell* hashArray = info.m_bodyAabbCell.m_array;

				unsigned count = 0;
				const cuBodyAabbCell& cell = hashArray[index];

				for (int i = index + 1; cell.m_key == hashArray[i].m_key; i++)
				{
					count++;
				}
				cacheBuffer[threadId1] = count;
			}

			__syncthreads();
			for (int i = 1; i < D_THREADS_PER_BLOCK; i = i << 1)
			{
				int sum = cacheBuffer[threadId1] + cacheBuffer[threadId1 - i];
				__syncthreads();
				cacheBuffer[threadId1] = sum;
				__syncthreads();
			}

			if (index < cellCount)
			{
				unsigned* scan = info.m_histogram.m_array;
				const unsigned prefixScanSuperBlockAlign = D_PREFIX_SCAN_PASSES * D_THREADS_PER_BLOCK;
				const unsigned offset = (cellCount + prefixScanSuperBlockAlign) & (-prefixScanSuperBlockAlign);
				scan[offset + index] = cacheBuffer[threadId1];
			}
		}
	};
#endif

	//auto SortKey = [] __device__(const unsigned& item)
	auto SortKey = [] __device__(const ndCudaBodyAabbCell& item)
	{
		const unsigned key = item.m_key;
		return key & 0xff;
	};

	ndCudaSceneInfo* const infoGpu = m_sceneInfoGpu;

#if 0
	//int threads = m_context->m_bodyBufferGpu.GetCount() - 1;
	//int bodyBlocksCount = (threads + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	//CudaInitBodyArray << <bodyBlocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (InitBodyArray, *infoGpu);
	//CudaMergeAabb << <1, D_THREADS_PER_BLOCK, 0, stream >> > (MergeAabb, *infoGpu);
	//CudaCountAabb << <bodyBlocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (CountAabb, *infoGpu);
	//CudaPrefixScan(m_context, D_THREADS_PER_BLOCK);
	//CudaValidateGridBuffer << <1, 1, 0, stream >> > (ValidateGridArray, *infoGpu);
	//CudaGenerateGridHash << <bodyBlocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (GenerateHashGrids, *infoGpu);
	CudaBodyAabbCellSortBuffer(m_context);
	dAssert(SanityCheckSortCells());

	//	int cellsBlocksCount = (m_context->m_bodyAabbCell.m_capacity + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	//	dAssert(cellsBlocksCount > 0);
	//	CudaCalculateBodyPairsCount << <cellsBlocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (CalculateBodyPairsCount, *infoGpu);
	////dAssert(SanityCheckPrefix());
	//
	//
	//	//auto GetKey____ = [] __device__(const unsigned& item)
	//	//{
	//	//	return 0;
	//	//};
	//	//XXXXXXX << <1, 1, 0, stream >> > (GetKey____);
#endif

	ndCudaScene << <1, 1, 0, m_solverComputeStream >> > (*infoGpu);
	//ndCudaSortGridArray << <1, 1, 0, m_solverComputeStream >> > (*infoGpu, SortKey, SortKey, SortKey, SortKey);
}