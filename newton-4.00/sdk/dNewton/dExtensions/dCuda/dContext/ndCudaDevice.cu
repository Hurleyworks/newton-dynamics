#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include "ndCudaStdafx.h"
#include "ndCudaUtils.h"
#include "ndCudaDevice.h"

ndCudaDevice::ndCudaDevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipGetDeviceProperties(&m_prop, 0);
	ndAssert(cudaStatus == hipSuccess);
	
	cudaStatus = hipSetDevice(0);
	ndAssert(cudaStatus == hipSuccess);
	if (cudaStatus != hipSuccess)
	{
		ndAssert(0);
	}
	
	cuTrace(("gpu: %s\n", m_prop.name));
	cuTrace(("compute capability: %d.%d\n", m_prop.major, m_prop.minor));
	
	cuTrace(("warp size: %d\n", m_prop.warpSize));
	cuTrace(("multiprocessors: %d\n", m_prop.multiProcessorCount));
	cuTrace(("threads per blocks %d\n", m_prop.maxThreadsPerBlock));
	cuTrace(("blocks per multiprocessors %d\n", m_prop.maxBlocksPerMultiProcessor));
	cuTrace(("memory bus with: %d bits\n", m_prop.memoryBusWidth));
	cuTrace(("memory: (mbytes) %d\n", m_prop.totalGlobalMem / (1024 * 1024)));

	m_workGroupSize = std::min(m_prop.maxThreadsPerBlock, 512);
	m_computeUnits = std::min(4 * m_prop.multiProcessorCount, 512);
	//m_computeUnits = std::min(m_prop.maxBlocksPerMultiProcessor * m_prop.multiProcessorCount / 4, 512);
	//m_maxBlocksPerKernel = m_prop.maxBlocksPerMultiProcessor * m_prop.multiProcessorCount;
}

ndCudaDevice::~ndCudaDevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	ndAssert(cudaStatus == hipSuccess);
	
	if (cudaStatus != hipSuccess)
	{
		ndAssert(0);
	}
}

void* ndCudaDevice::operator new (size_t size)
{											
	return ndCudaMalloc(size);
}											

void ndCudaDevice::operator delete (void* ptr)
{											
	ndCudaFree(ptr);
}											

int ndCudaDevice::GetComputeUnits() const
{
	return m_computeUnits;
}

ndKernelParams::ndKernelParams(const ndCudaDevice* const device, int workGroupSize, int itemCount)
	:m_itemCount(itemCount)
	,m_workGroupSize(workGroupSize)
{
	ndAssert(workGroupSize);
	ndAssert(!(workGroupSize & (workGroupSize - 1)));
	int deviceComputeUnits = device->GetComputeUnits();
	int computeUnitsBashCount = (itemCount + m_workGroupSize - 1) / m_workGroupSize;

	m_blocksPerKernel = (computeUnitsBashCount + deviceComputeUnits - 1) / deviceComputeUnits;
	m_kernelCount = (itemCount + m_blocksPerKernel * m_workGroupSize - 1) / (m_blocksPerKernel * m_workGroupSize);
	ndAssert(m_kernelCount <= deviceComputeUnits);
}
