/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include "ndCudaContext.h"

ndCudaContext::ndCudaContext()
	:ndClassAlloc()
	,m_bodyBuffer()
{
	hipError_t cudaStatus;
	cudaStatus = hipGetDeviceProperties(&m_prop, 0);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipSetDevice(0);
	dAssert(cudaStatus == hipSuccess);
	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

ndCudaContext::~ndCudaContext()
{
}

ndCudaContext* ndCudaContext::CreateContext()
{
	hipError_t cudaStatus = hipSetDevice(0);
	return (cudaStatus == hipSuccess) ? new ndCudaContext() : nullptr;
}
