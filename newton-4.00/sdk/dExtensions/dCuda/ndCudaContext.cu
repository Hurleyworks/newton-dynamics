/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include "ndCudaContext.h"

ndCudaContext::ndCudaContext()
	:ndClassAlloc()
	,m_bodyBufferCpu(D_GRANULARITY)
	,m_bodyBufferGpu()
	,m_transformBufferCpu0()
	,m_transformBufferCpu1()
	,m_transformBufferGpu()
	,m_stream0(0)
{
	hipError_t cudaStatus;
	cudaStatus = hipGetDeviceProperties(&m_prop, 0);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipSetDevice(0);
	dAssert(cudaStatus == hipSuccess);

	// create tow strem for double buffer updates
	cudaStatus = hipStreamCreate(&m_stream0);
	dAssert(cudaStatus == hipSuccess);

	//cudaStatus = hipStreamCreate(&m_stream1);
	//dAssert(cudaStatus == hipSuccess);

	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

ndCudaContext::~ndCudaContext()
{
	hipError_t cudaStatus;
	cudaStatus = hipStreamDestroy(m_stream0);
	dAssert(cudaStatus == hipSuccess);

	//cudaStatus = hipStreamDestroy(m_stream1);
	//dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipDeviceReset();
	dAssert(cudaStatus == hipSuccess);

	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

ndCudaContext* ndCudaContext::CreateContext()
{
	hipError_t cudaStatus = hipSetDevice(0);
	return (cudaStatus == hipSuccess) ? new ndCudaContext() : nullptr;
}

void ndCudaContext::SwapBuffers()
{
	//dSwap(m_stream0, m_stream1);
	m_transformBufferCpu0.Swap(m_transformBufferCpu1);
}