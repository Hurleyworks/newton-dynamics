/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include "ndCudaContext.h"

ndCudaContext::ndCudaContext()
	:ndClassAlloc()
	,m_bodyBuffer()
{
	hipError_t cudaStatus;
	cudaStatus = hipGetDeviceProperties(&m_prop, 0);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipSetDevice(0);
	dAssert(cudaStatus == hipSuccess);
	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

ndCudaContext::~ndCudaContext()
{
}

ndCudaContext* ndCudaContext::CreateContext()
{
	//cl_uint numPlatforms = 0;
	//cl_int err = clGetPlatformIDs(0, nullptr, &numPlatforms);
	//if ((err != CL_SUCCESS) || (numPlatforms == 0))
	//{
	//	return nullptr;
	//}
	//
	//dAssert(numPlatforms < 16);
	//cl_platform_id platforms[16];
	//err = clGetPlatformIDs(numPlatforms, &platforms[0], nullptr);
	//if (err != CL_SUCCESS)
	//{
	//	return nullptr;
	//}
	//
	//ndInt32 driveIndex = 0;
	//cl_platform_id bestPlatform = 0;
	//for (cl_uint i = 0; i < numPlatforms; i++)
	//{
	//	cl_uint numDevices = 0;
	//	err = clGetDeviceIDs(platforms[i], CL_DEVICE_TYPE_GPU, 0, nullptr, &numDevices);
	//	if (!((err != CL_SUCCESS) || (numDevices == 0)))
	//	{
	//		bestPlatform = platforms[i];
	//		if (driveIndex == driveNumber)
	//		{
	//			break;
	//		}
	//		driveIndex++;
	//	}
	//}
	//
	//if (bestPlatform == nullptr)
	//{
	//	return nullptr;
	//}
	//
	//cl_context_properties contextProperties[] = { CL_CONTEXT_PLATFORM, (cl_context_properties)bestPlatform, 0 };
	//
	//cl_context context = clCreateContextFromType(contextProperties, CL_DEVICE_TYPE_GPU, nullptr, nullptr, &err);
	//if ((CL_SUCCESS != err) || (context == nullptr))
	//{
	//	return nullptr;
	//}

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		return nullptr;
	}

	return new ndCudaContext();
}
