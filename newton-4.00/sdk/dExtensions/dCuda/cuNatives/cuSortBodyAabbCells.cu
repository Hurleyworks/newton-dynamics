#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <ndNewtonStdafx.h>

#include "cuIntrisics.h"
#include "ndCudaContext.h"
#include "cuSortBodyAabbCells.h"

#define D_PARALLEL_PREFIX_LOCAL_SIZE	(1024)
#define D_PARALLEL_PREFIX_BUFFER_SIZE	(D_PARALLEL_PREFIX_LOCAL_SIZE * 4)


//__global__ void cuTest0(const cuSceneInfo& info, int digit)
//{
//
//}
//
//__global__ void cuTest1(const cuSceneInfo& info, int digit, hipStream_t stream)
//{
//	cuTest0 << <1, 1, 0, stream >> > (info, digit);
//}

//void BitonicSort(int* arr, int D_THREADS_PER_BLOCK)
//{
//	for (int i = 0; i < D_THREADS_PER_BLOCK * 4; i++)
//	{
//		arr[i] = (D_THREADS_PER_BLOCK << D_THREADS_PER_BLOCK_BITS) + i;
//	}
//
//	arr[0] = (0 << D_THREADS_PER_BLOCK_BITS) + 0;
//	arr[1] = (1 << D_THREADS_PER_BLOCK_BITS) + 1;
//	arr[2] = (0 << D_THREADS_PER_BLOCK_BITS) + 2;
//	arr[3] = (1 << D_THREADS_PER_BLOCK_BITS) + 3;
//
//	int xxxx = 0;
//	for (int k = 2; k <= D_THREADS_PER_BLOCK; k *= 2)
//	{
//		for (int j = k / 2; j > 0; j /= 2)
//		{
//			xxxx++;
//			for (int i = 0; i < D_THREADS_PER_BLOCK; i++)
//			{
//				const int l = i ^ j;
//				if (l > i)
//				{
//					const int mask0 = (-(i & k)) >> 31;
//					const int mask1 = (-(arr[i] > arr[l])) >> 31;
//					const int mask = mask0 ^ mask1;
//					const int a = arr[i];
//					const int b = arr[l];
//					arr[i] = b & mask | a & ~mask;
//					arr[l] = a & mask | b & ~mask;
//				}
//			}
//		}
//	}
//	arr[D_THREADS_PER_BLOCK - 1] = 0;
//}

inline bool __device__ cuCountingSortIsThisGridCellDigitValid(const cuSceneInfo& info, int digit)
{
	bool isEven = (digit & 1) ? false : true;
	bool hasUpperByteHash = (&info.m_hasUpperByteHash.x)[digit / 4] ? true : false;
	bool test = isEven | hasUpperByteHash;
	return test;
}

inline unsigned __device__ cuCountingSortEvaluateGridCellKey(const cuBodyAabbCell& dataElement, int digit)
{
	return dataElement.m_bytes[digit];
};

__global__ void cuCountingSortCountGridCells(const cuSceneInfo& info, int digit)
{
	__shared__  int cacheBuffer[D_THREADS_PER_BLOCK];
	if (info.m_frameIsValid)
	{
		bool test = cuCountingSortIsThisGridCellDigitValid(info, digit);
		if (test)
		{
			const int cellCount = info.m_bodyAabbCell.m_size - 1;
			const int blocks = (cellCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
			if (blockIdx.x < blocks)
			{
				int threadId = threadIdx.x;
				cacheBuffer[threadId] = 0;

				int index = threadId + blockDim.x * blockIdx.x;

				int* histogram = info.m_histogram.m_array;
				const cuBodyAabbCell* src = (digit & 1) ? info.m_bodyAabbCellScrath.m_array : info.m_bodyAabbCell.m_array;

				__syncthreads();
				if (index < cellCount)
				{
					unsigned key = cuCountingSortEvaluateGridCellKey(src[index], digit);
					atomicAdd(&cacheBuffer[key], 1);
				}
				__syncthreads();
				histogram[index] = cacheBuffer[threadId];
			}
		}
	}
}

__global__ void cuCountingSortShuffleGridCells(const cuSceneInfo& info, int digit)
{
	if (info.m_frameIsValid)
	{
		const int threadId = threadIdx.x;
		const int index = threadId + blockDim.x * blockIdx.x;
		const int cellCount = info.m_bodyAabbCell.m_size - 1;
		const bool test = cuCountingSortIsThisGridCellDigitValid(info, digit);
		if (test)
		{
			const int blocks = (cellCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
#if 0
			__shared__  int cacheKey[D_THREADS_PER_BLOCK];
			__shared__  int cacheBufferCount[D_THREADS_PER_BLOCK];
			__shared__  int cacheBufferAdress[D_THREADS_PER_BLOCK];
			if (blockIdx.x < blocks)
			{
				int* histogram = info.m_histogram.m_array;
				cuBodyAabbCell* dst = (digit & 1) ? info.m_bodyAabbCell.m_array : info.m_bodyAabbCellScrath.m_array;
				const cuBodyAabbCell* src = (digit & 1) ? info.m_bodyAabbCellScrath.m_array : info.m_bodyAabbCell.m_array;
			
				cacheKey[threadId] = 0;
				cacheBufferCount[threadId] = histogram[index];
				if (index < cellCount)
				{
					const cuBodyAabbCell entry = src[index];
					cacheKey[threadId] = cuCountingSortEvaluateGridCellKey(entry, digit);
					__syncthreads();
			
					if (threadId == 0)
					{
						for (int i = 0; i < D_THREADS_PER_BLOCK; i++)
						{
							const int key = cacheKey[i];
							const int dstIndex = cacheBufferCount[key];
							cacheBufferAdress[i] = dstIndex;
							cacheBufferCount[key] = dstIndex + 1;
						}
					}
					__syncthreads();
					int dstIndex = cacheBufferAdress[threadId];
					dst[dstIndex] = entry;
				}
			}

#else
			__shared__  int cacheSortedKey[D_THREADS_PER_BLOCK];
			__shared__  int cacheBufferCount[D_THREADS_PER_BLOCK];
			__shared__  int cacheBufferAdress[2 * D_THREADS_PER_BLOCK];
			__shared__  int cacheKeyPrefix[D_THREADS_PER_BLOCK / 2 + D_THREADS_PER_BLOCK + 1];

			if (blockIdx.x < blocks)
			{
				int* histogram = info.m_histogram.m_array;
				cuBodyAabbCell* dst = (digit & 1) ? info.m_bodyAabbCell.m_array : info.m_bodyAabbCellScrath.m_array;
				const cuBodyAabbCell* src = (digit & 1) ? info.m_bodyAabbCellScrath.m_array : info.m_bodyAabbCell.m_array;

				cacheKeyPrefix[threadId] = 0;
				cacheKeyPrefix[threadId + D_THREADS_PER_BLOCK/2 + 1] = 0;
				cacheBufferCount[threadId] = histogram[index];
				cacheSortedKey[threadId] = (D_THREADS_PER_BLOCK << D_THREADS_PER_BLOCK_BITS) | threadId;

				cuBodyAabbCell entry;
				if (index < cellCount)
				{
					entry = src[index];
					const int key = cuCountingSortEvaluateGridCellKey(entry, digit);
					atomicAdd(&cacheKeyPrefix[key + D_THREADS_PER_BLOCK/2 + 1], 1);
					cacheSortedKey[threadId] = (key << D_THREADS_PER_BLOCK_BITS) | threadId;
				}
				__syncthreads();

				for (int k = 2; k <= D_THREADS_PER_BLOCK; k *= 2)
				{
					for (int j = k / 2; j > 0; j /= 2)
					{
						const int threadId1 = threadId ^ j;
						if (threadId1 > threadId)
						{
							const int a = cacheSortedKey[threadId];
							const int b = cacheSortedKey[threadId1];
							const int mask0 = (-(threadId & k)) >> 31;
							const int mask1 = (-(a > b)) >> 31;
							const int mask = mask0 ^ mask1;
							cacheSortedKey[threadId] = b & mask | a & ~mask;
							cacheSortedKey[threadId1] = a & mask | b & ~mask;
						}
						__syncthreads();
					}
				}

				const int threadId1 = threadId + D_THREADS_PER_BLOCK/2;
				for (int i = 1; i < D_THREADS_PER_BLOCK; i = i << 1)
				{
					int sum = cacheKeyPrefix[threadId1] + cacheKeyPrefix[threadId1 - i];
					__syncthreads();
					cacheKeyPrefix[threadId1] = sum;
					__syncthreads();
				}

				const int itemCount = cuMin(int(cellCount - blockIdx.x * D_THREADS_PER_BLOCK), D_THREADS_PER_BLOCK);
				for (int i = D_THREADS_PER_BLOCK/2; cacheKeyPrefix[i] < itemCount; i++)
				{
					const int start = cacheKeyPrefix[i];
					const int count = cacheKeyPrefix[i + 1] - start;
					if (threadId < count)
					{
						const int key = cacheSortedKey[start] >> D_THREADS_PER_BLOCK_BITS;
						const int dstIndex = cacheSortedKey[start + threadId] & (D_THREADS_PER_BLOCK - 1);
						const int base = cacheBufferCount[key];
						cacheBufferAdress[dstIndex] = base + threadId;
					}
				}
				__syncthreads();

				if (index < cellCount)
				{
					int dstIndex = cacheBufferAdress[threadId];
					dst[dstIndex] = entry;
				}
			}
#endif
		}
		else
		{
			if (index < cellCount)
			{
				cuBodyAabbCell* dst = info.m_bodyAabbCell.m_array;
				const cuBodyAabbCell* src = info.m_bodyAabbCellScrath.m_array;
				dst[index] = src[index];
			}
		}
	}
}

__global__ void cuCountingSortClearLastSuperBlock(const cuSceneInfo& info, int digit)
{
	if (info.m_frameIsValid)
	{
		bool test = cuCountingSortIsThisGridCellDigitValid(info, digit);
		if (test)
		{
			const int cellCount = info.m_bodyAabbCell.m_size - 1;
			const int blocks = (cellCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
			const int superBlocks = (blocks + D_COUNT_SORT_SUPER_BLOCK - 1) / D_COUNT_SORT_SUPER_BLOCK;

			const int offset = threadIdx.x + (superBlocks - 1) * D_COUNT_SORT_SUPER_BLOCK * D_THREADS_PER_BLOCK;
			int* histogram = info.m_histogram.m_array;
			for (int i = 0; i < D_COUNT_SORT_SUPER_BLOCK; i++)
			{
				histogram[offset + i * D_THREADS_PER_BLOCK] = 0;
			}
		}
	}
}

__global__ void cuCountingSortAddSuperBlock(const cuSceneInfo& info, int digit)
{
	if (info.m_frameIsValid)
	{
		bool test = cuCountingSortIsThisGridCellDigitValid(info, digit);
		if (test)
		{
			const int cellCount = info.m_bodyAabbCell.m_size - 1;
			const int blocks = (cellCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
			const int superBlocks = (blocks + D_COUNT_SORT_SUPER_BLOCK - 1) / D_COUNT_SORT_SUPER_BLOCK;
			if (blockIdx.x < superBlocks)
			{
				const int threadId = threadIdx.x;
				const int superBlockSize = D_COUNT_SORT_SUPER_BLOCK * D_THREADS_PER_BLOCK;
				const int offsetIn = threadId + blockIdx.x * superBlockSize;
				int* histogram = info.m_histogram.m_array;

				int sum = 0;
				for (int i = 0; i < D_COUNT_SORT_SUPER_BLOCK; i++)
				{
					sum += histogram[offsetIn + i * D_THREADS_PER_BLOCK];
				}
				const int offset = threadId + superBlocks * superBlockSize + blockIdx.x * D_THREADS_PER_BLOCK;
				histogram[offset] = sum;
			}
		}
	}
}

__global__ void cuCountingSortBodyCellsPrefixScan(const cuSceneInfo& info, int digit)
{
	__shared__  int cacheBuffer[D_THREADS_PER_BLOCK / 2 + D_THREADS_PER_BLOCK + 1];

	if (info.m_frameIsValid)
	{
		bool test = cuCountingSortIsThisGridCellDigitValid(info, digit);
		if (test)
		{
			const int threadId = threadIdx.x;
			cacheBuffer[threadId] = 0;
			cacheBuffer[threadId + D_THREADS_PER_BLOCK / 2 + 1] = 0;

			int* histogram = info.m_histogram.m_array;
			const int cellCount = info.m_bodyAabbCell.m_size - 1;
			const int blocks = (cellCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
			const int superBlocks = (blocks + D_COUNT_SORT_SUPER_BLOCK - 1) / D_COUNT_SORT_SUPER_BLOCK;
			const int superBlockOffset = threadId + superBlocks * D_COUNT_SORT_SUPER_BLOCK * D_THREADS_PER_BLOCK;

			int sum = 0;
			const int threadId1 = threadId + D_THREADS_PER_BLOCK / 2;
			for (int i = 0; i < superBlocks; i++)
			{
				sum += histogram[superBlockOffset + i * D_THREADS_PER_BLOCK];
			}
			cacheBuffer[threadId1 + 1] = sum;
			__syncthreads();

			for (int i = 1; i < D_THREADS_PER_BLOCK; i = i << 1)
			{
				const int prefixSum = cacheBuffer[threadId1] + cacheBuffer[threadId1 - i];
				__syncthreads();
				cacheBuffer[threadId1] = prefixSum;
				__syncthreads();
			}

			sum = cacheBuffer[threadId1];
			for (int i = 0; i < blocks; i++)
			{
				int j = i * D_THREADS_PER_BLOCK + threadId;
				int partialSum = histogram[j];
				histogram[j] = sum;
				sum += partialSum;
			}
		}
	}
}


static bool CountingSortBodyCellsSanityCheck(ndCudaContext* const context)
{
	cuSceneInfo info;
	hipError_t cudaStatus;
	hipDeviceSynchronize();
	cudaStatus = hipMemcpy(&info, context->m_sceneInfoGpu, sizeof(cuSceneInfo), hipMemcpyDeviceToHost);
	dAssert(cudaStatus == hipSuccess);

	if (info.m_frameIsValid)
	{
		static ndArray<cuBodyAabbCell> data;
		int size = info.m_bodyAabbCell.m_size - 1;
		data.SetCount(size);
		cudaStatus = hipMemcpy(&data[0], info.m_bodyAabbCell.m_array, size * sizeof(cuBodyAabbCell), hipMemcpyDeviceToHost);
		//cudaStatus = hipMemcpy(&data[0], info.m_bodyAabbCellScrath.m_array, size * sizeof(cuBodyAabbCell), hipMemcpyDeviceToHost);
		dAssert(cudaStatus == hipSuccess);

		for (int i = 1; i < size; i++)
		{
			cuBodyAabbCell key0(data[i - 1]);
			cuBodyAabbCell key1(data[i - 0]);
			bool zTest0 = key0.m_z < key1.m_z;
			bool zTest1 = key0.m_z == key1.m_z;
			bool yTest0 = key0.m_y < key1.m_y;
			bool yTest1 = key0.m_y == key1.m_y;
			bool xTest = key0.m_x <= key1.m_x;
			bool test = zTest0 | (zTest1 & (yTest0 | (yTest1 & xTest)));
			//test = xTest;
			//test = key0.m_y <= key1.m_y;
			//test = yTest0 | (yTest1 & xTest);
			dAssert(test);
		}
	}
	return true;
}

static void CountingSortBodyCells(ndCudaContext* context, int digit)
{
	cuSceneInfo* const sceneInfo = context->m_sceneInfoCpu;
	ndInt32 blocks = (sceneInfo->m_bodyAabbCell.m_capacity + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	if (blocks)
	{
		hipStream_t stream = context->m_solverComputeStream;
		cuSceneInfo* const infoGpu = context->m_sceneInfoGpu;
		ndInt32 superBlocks = (blocks + D_COUNT_SORT_SUPER_BLOCK - 1) / D_COUNT_SORT_SUPER_BLOCK;

		cuCountingSortClearLastSuperBlock << <1, D_THREADS_PER_BLOCK, 0, stream >> > (*infoGpu, digit);
		cuCountingSortCountGridCells << <blocks, D_THREADS_PER_BLOCK, 0, stream >> > (*infoGpu, digit);
		cuCountingSortAddSuperBlock << <superBlocks, D_THREADS_PER_BLOCK, 0, stream >> > (*infoGpu, digit);
		cuCountingSortBodyCellsPrefixScan << <1, D_THREADS_PER_BLOCK, 0, stream >> > (*infoGpu, digit);
		cuCountingSortShuffleGridCells << <blocks, D_THREADS_PER_BLOCK, 0, stream >> > (*infoGpu, digit);
	}
}

void CudaSortBodyAabbCells(ndCudaContext* const context)
{
	//BitonicSort();

	dAssert(context->m_bodyAabbCell.GetCount() <= context->m_histogram.GetCount());
	dAssert(context->m_bodyAabbCell.GetCount() == context->m_bodyAabbCellTmp.GetCount());

	for (int i = 0; i < 3; i++)
	{
		CountingSortBodyCells(context, i * 4 + 0);
		CountingSortBodyCells(context, i * 4 + 1);
	}

	dAssert(CountingSortBodyCellsSanityCheck(context));
}
