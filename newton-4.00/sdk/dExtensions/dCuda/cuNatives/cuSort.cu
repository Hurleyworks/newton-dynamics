#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <ndNewtonStdafx.h>
#include "cuSort.h"
#include "ndCudaContext.h"


//__global__ void cuTest0(const cuSceneInfo& info, int digit)
//{
//
//}
//
//__global__ void cuTest1(const cuSceneInfo& info, int digit, hipStream_t stream)
//{
//	cuTest0 << <1, 1, 0, stream >> > (info, digit);
//}



inline bool __device__ cuIsThisGridHashDigitValid(const cuSceneInfo& info, int digit)
{
	bool isEven = (digit & 1) ? false : true;
	bool hasUpperByteHash = (&info.m_hasUpperByteHash.x)[digit / 4] ? true : false;
	bool test = isEven | hasUpperByteHash;
	return test;
}

inline unsigned __device__ cuEvaluateGridHashKey(const cuAabbGridHash& dataElement, int digit)
{
	return dataElement.m_bytes[digit];
};

__global__ void cuGridHashPrefixScan(const cuSceneInfo& info, int digit)
{
	__shared__  int cacheBuffer[2 * D_THREADS_PER_BLOCK + 1];

	if (info.m_frameIsValid)
	{
		bool test = cuIsThisGridHashDigitValid(info, digit);
		if (test)
		{
			int threadId = threadIdx.x;
			int threadId1 = threadId + D_THREADS_PER_BLOCK;

			int sum = 0;
			cacheBuffer[threadId] = 0;
			if (threadId == 0)
			{
				cacheBuffer[threadId1] = 0;
			}

			int* histogram = info.m_histogram.m_array;
			const int blocks = (info.m_histogram.m_size + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
			for (int i = 0; i < blocks; i++)
			{
				sum += histogram[i * D_THREADS_PER_BLOCK + threadId];
			}
			cacheBuffer[threadId1 + 1] = sum;
			__syncthreads();

			for (int i = 1; i < D_THREADS_PER_BLOCK; i = i << 1)
			{
				int sum = cacheBuffer[threadId1] + cacheBuffer[threadId1 - i];
				__syncthreads();
				cacheBuffer[threadId1] = sum;
				__syncthreads();
			}
			sum = cacheBuffer[threadId1];

			for (int i = 0; i < blocks; i++)
			{
				int j = i * D_THREADS_PER_BLOCK + threadId;
				int partialSum = histogram[j];
				histogram[j] = sum;
				sum += partialSum;
			}
		}
	}
}

__global__ void cuCountGridHashKeys(const cuSceneInfo& info, int digit)
{
	__shared__  int cacheBuffer[D_THREADS_PER_BLOCK];
	if (info.m_frameIsValid)
	{
		bool test = cuIsThisGridHashDigitValid(info, digit);
		if (test)
		{
			int threadIndex = threadIdx.x;
			cacheBuffer[threadIndex] = 0;

			int index = threadIndex + blockDim.x * blockIdx.x;

			int* histogram = info.m_histogram.m_array;
			const cuAabbGridHash* src = (digit & 1) ? info.m_hashArrayScrath.m_array : info.m_hashArray.m_array;
		
			__syncthreads();
			int gridCount = info.m_hashArray.m_size - 1;
			if (index < gridCount)
			{
				unsigned key = cuEvaluateGridHashKey(src[index], digit);
				atomicAdd(&cacheBuffer[key], 1);
			}
			__syncthreads();
			histogram[index] = cacheBuffer[threadIndex];
		}
	}
}

__global__ void cuSortGridHashItems(const cuSceneInfo& info, int digit)
{
	__shared__  int cacheKey[D_THREADS_PER_BLOCK];
	__shared__  int cacheBufferCount[D_THREADS_PER_BLOCK];
	__shared__  int cacheBufferAdress[D_THREADS_PER_BLOCK];

	if (info.m_frameIsValid)
	{
		int threadIndex = threadIdx.x;
		int index = threadIndex + blockDim.x * blockIdx.x;
		const int gridCount = info.m_hashArray.m_size - 1;
		bool test = cuIsThisGridHashDigitValid(info, digit);
		if (test)
		{
			int* histogram = info.m_histogram.m_array;
			cuAabbGridHash* dst = (digit & 1) ? info.m_hashArray.m_array: info.m_hashArrayScrath.m_array;
			const cuAabbGridHash* src = (digit & 1) ? info.m_hashArrayScrath.m_array : info.m_hashArray.m_array;

			cacheKey[threadIndex] = 0;
			if (index < gridCount)
			{
				const cuAabbGridHash entry = src[index];
				cacheBufferCount[threadIndex] = histogram[index];
				cacheKey[threadIndex] = cuEvaluateGridHashKey(entry, digit);
				__syncthreads();
				
				if (threadIndex == 0)
				{
					for (int i = 0; i < D_THREADS_PER_BLOCK; i++)
					{
						const int key = cacheKey[i];
						const int dstIndex = cacheBufferCount[key];
						cacheBufferAdress[i] = dstIndex;
						cacheBufferCount[key] = dstIndex + 1;
					}
				}
				__syncthreads();
				int dstIndex = cacheBufferAdress[threadIndex];
				dst[dstIndex] = entry;
			}
		}
		else
		{
			if (index < gridCount)
			{
				cuAabbGridHash* dst = info.m_hashArray.m_array;
				const cuAabbGridHash* src = info.m_hashArrayScrath.m_array;
				dst[index] = src[index];
			}
		}
	}
}

static bool GridHashSanityCheck(ndCudaContext* const context)
{
	cuSceneInfo info;
	hipError_t cudaStatus;
	hipDeviceSynchronize();
	cudaStatus = hipMemcpy(&info, context->m_sceneInfoGpu, sizeof(cuSceneInfo), hipMemcpyDeviceToHost);
	dAssert(cudaStatus == hipSuccess);

	if (info.m_frameIsValid)
	{
		ndArray<cuAabbGridHash> data;
		int size = info.m_hashArray.m_size;
		data.SetCount(size);
		cudaStatus = hipMemcpy(&data[0], info.m_hashArray.m_array, size * sizeof(cuAabbGridHash), hipMemcpyDeviceToHost);
		dAssert(cudaStatus == hipSuccess);

		for (int i = 1; i < size; i++)
		{
			cuAabbGridHash key0(data[i - 1]);
			cuAabbGridHash key1(data[i - 0]);
			bool zTest0 = key0.m_z < key1.m_z;
			bool zTest1 = key0.m_z == key1.m_z;
			bool yTest0 = key0.m_y < key1.m_y;
			bool yTest1 = key0.m_y == key1.m_y;
			bool xTest = key0.m_x <= key1.m_x;
			//bool test = zTest0 | (zTest1 & (yTest0 | (yTest1 & xTest)));
			bool test = xTest;
			dAssert(test);
		}
	}
	return true;
}

static void SortGridHash(ndCudaContext* context, int digit)
{
	hipStream_t stream = context->m_stream0;
	cuSceneInfo* const infoGpu = context->m_sceneInfoGpu;
	cuSceneInfo* const sceneInfo = context->m_sceneInfoCpu;
	ndInt32 blocks = (sceneInfo->m_histogram.m_size + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	if (blocks)
	{
		cuCountGridHashKeys << <blocks, D_THREADS_PER_BLOCK, 0, stream >> > (*infoGpu, digit);
		cuGridHashPrefixScan << <1, D_THREADS_PER_BLOCK, 0, stream >> > (*infoGpu, digit);
		cuSortGridHashItems << <blocks, D_THREADS_PER_BLOCK, 0, stream >> > (*infoGpu, digit);
	}
}

void CudaSortGridHash(ndCudaContext* const context)
{
	dAssert(context->m_gridHash.GetCount() == context->m_gridHashTmp.GetCount());
	dAssert(context->m_histogram.GetCount() == context->m_gridHashTmp.GetCount());

	for (int i = 0; i < 1; i++)
	{
		SortGridHash(context, i * 4 + 0);
		SortGridHash(context, i * 4 + 1);
	}
	dAssert(GridHashSanityCheck(context));
}
