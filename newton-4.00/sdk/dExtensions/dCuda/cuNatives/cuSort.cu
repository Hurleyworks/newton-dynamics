#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <ndNewtonStdafx.h>
#include "cuSort.h"
#include "ndCudaContext.h"

inline bool __device__ HasUpperDigit(const cuSceneInfo& info, int digit)
{
	bool hasUpperByteHash = (&info.m_hasUpperByteHash.x)[digit / 4] ? true : false;
	bool test = !(digit & 1) | hasUpperByteHash;
	return test;
}

template <typename Predicate>
__global__ void CudaSortHistogram(Predicate EvaluateKey, const cuSceneInfo& info, const cuAabbGridHash* src, int* histogram, int size, int digit)
{
	__shared__  int cacheBuffer[D_THREADS_PER_BLOCK];

	bool test = HasUpperDigit(info, digit);
	if (test)
	{
		int threadIndex = threadIdx.x;
		int index = threadIndex + blockDim.x * blockIdx.x;

		cacheBuffer[threadIndex] = 0;
		__syncthreads();
		if (index < size)
		{
			int key = EvaluateKey(src[index], digit);
			atomicAdd(&cacheBuffer[key], 1);
		}
		__syncthreads();
		histogram[index] = cacheBuffer[threadIndex];
	}
}

template <typename Predicate>
__global__ void CudaSortItems(Predicate EvaluateKey, const cuSceneInfo& info, const cuAabbGridHash* src, cuAabbGridHash* dst, int* histogram, int size, int digit)
{
	__shared__  int cacheKey[D_THREADS_PER_BLOCK];
	__shared__  int cacheBufferCount[D_THREADS_PER_BLOCK];
	__shared__  int cacheBufferAdress[D_THREADS_PER_BLOCK];

	int threadIndex = threadIdx.x;
	int index = threadIndex + blockDim.x * blockIdx.x;

	cacheKey[threadIndex] = 0;
	if (index < size)
	{
		bool test = HasUpperDigit(info, digit);
		if (test)
		{
			const cuAabbGridHash entry = src[index];
			cacheBufferCount[threadIndex] = histogram[index];
			cacheKey[threadIndex] = EvaluateKey(entry, digit);
			__syncthreads();

			if (threadIndex == 0)
			{
				for (int i = 0; i < D_THREADS_PER_BLOCK; i++)
				{
					const int key = cacheKey[i];
					const int dstIndex = cacheBufferCount[key];
					cacheBufferAdress[i] = dstIndex;
					cacheBufferCount[key] = dstIndex + 1;
				}
			}
			__syncthreads();
			int dstIndex = cacheBufferAdress[threadIndex];
			dst[dstIndex] = entry;
		}
		else
		{
			dst[index] = src[index];
		}
	}
}

template <typename Predicate>
__global__ void CudaSortPrefixScans(Predicate PrefixScan, const cuSceneInfo& info, int* histogram, int size, int digit)
{
	bool test = HasUpperDigit(info, digit);
	if (test)
	{
		PrefixScan(histogram, size);
	}
}

CudaCountingSort::CudaCountingSort(cuSceneInfo* info, hipStream_t stream)
	:m_info(info)
	,m_stream(stream)
{
}

bool CudaCountingSort::SanityCheck(const cuAabbGridHash* const src)
{
	//cuSceneInfo info;
	//hipError_t cudaStatus;
	//ndArray<cuAabbGridHash> data;
	//
	//hipDeviceSynchronize();
	//cudaStatus = hipMemcpy(&info, m_info, sizeof(cuSceneInfo), hipMemcpyDeviceToHost);
	//dAssert(cudaStatus == hipSuccess);
	//
	//data.SetCount(m_size);
	//cudaStatus = hipMemcpy(&data[0], src, m_size * sizeof(cuAabbGridHash), hipMemcpyDeviceToHost);
	//dAssert(cudaStatus == hipSuccess);
	//
	//for (int i = 1; i < m_size; i++)
	//{
	//	cuAabbGridHash key0(data[i - 1]);
	//	cuAabbGridHash key1(data[i - 0]);
	//	bool zTest0 = key0.m_z < key1.m_z;
	//	bool zTest1 = key0.m_z == key1.m_z;
	//	bool yTest0 = key0.m_y < key1.m_y;
	//	bool yTest1 = key0.m_y == key1.m_y;
	//	bool xTest = key0.m_x <= key1.m_x;
	//	bool test = zTest0 | (zTest1 & (yTest0 | (yTest1 & xTest)));
	//	dAssert(test);
	//}
	return true;
}

void CudaCountingSort::Sort(const cuAabbGridHash* const src, cuAabbGridHash* const dst, int digit)
{
	auto EvaluateKey = [] __device__(const cuAabbGridHash & dataElement, int digit)
	{
		return dataElement.m_bytes[digit];
	};

	auto PrefixScanSum = [] __device__(int* histogram, int size)
	{
		__shared__  int cacheBuffer[2 * D_THREADS_PER_BLOCK + 1];

		int threadId = threadIdx.x;
		int threadId1 = threadId + D_THREADS_PER_BLOCK;

		int sum = 0;
		cacheBuffer[threadId] = 0;
		if (threadId == 0)
		{
			cacheBuffer[threadId1] = 0;
		}
		const int blocks = size / D_THREADS_PER_BLOCK;
		for (int i = 0; i < blocks; i++)
		{
			sum += histogram[i * D_THREADS_PER_BLOCK + threadId];
		}
		cacheBuffer[threadId1 + 1] = sum;
		__syncthreads();

		for (int i = 1; i < D_THREADS_PER_BLOCK; i = i << 1)
		{
			int sum = cacheBuffer[threadId1] + cacheBuffer[threadId1 - i];
			__syncthreads();
			cacheBuffer[threadId1] = sum;
			__syncthreads();
		}
		sum = cacheBuffer[threadId1];

		for (int i = 0; i < blocks; i++)
		{
			int j = i * D_THREADS_PER_BLOCK + threadId;
			int partialSum = histogram[j];
			histogram[j] = sum;
			sum += partialSum;
		}
	};

	//m_size(size)
	//m_blocks((m_size + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK)
	//ndInt32 hashBlocksCount = (infoGpu->m_scan.m_size + 8 * D_THREADS_PER_BLOCK) / D_THREADS_PER_BLOCK; 
	 
	//CudaSortHistogram << <m_blocks, D_THREADS_PER_BLOCK, 0, m_stream >> > (EvaluateKey, *m_info, src, m_histogram, m_size, digit);
	//CudaSortPrefixScans << <1, D_THREADS_PER_BLOCK, 0, m_stream >> > (PrefixScanSum, *m_info, m_histogram, m_size, digit);
	//CudaSortItems << <m_blocks, D_THREADS_PER_BLOCK, 0, m_stream >> > (EvaluateKey, *m_info, src, dst, m_histogram, m_size, digit);
}

void CudaCountingSort::Sort()
{
	//cuAabbGridHash* const src, cuAabbGridHash* const dst
	//for (int i = 0; i < 3; i++)
	//{
	//	Sort(src, dst, i * 4 + 0);
	//	Sort(dst, src, i * 4 + 1);
	//}
	//dAssert(SanityCheck(src));
}
