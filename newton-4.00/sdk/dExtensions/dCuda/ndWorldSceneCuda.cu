#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include <ndWorld.h>
#include <ndModel.h>
#include <ndWorldScene.h>
#include <ndBodyDynamic.h>
#include <ndSkeletonList.h>
#include <ndDynamicsUpdate.h>
#include <ndBodyParticleSet.h>
#include <ndDynamicsUpdateSoa.h>
#include <ndJointBilateralConstraint.h>

#include "cuQuat.h"
#include "cuVector.h"
#include "cuMatrix3x3.h"

#include "ndCudaContext.h"
#include "ndWorldSceneCuda.h"

#define D_CUDA_SCENE_GRID_SIZE		8.0f
#define D_CUDA_SCENE_INV_GRID_SIZE	(1.0f/D_CUDA_SCENE_GRID_SIZE) 

template <typename Predicate>
__global__ void CudaAddBodyPadding(Predicate PaddLastBlock, cuBodyProxy* bodyArray, int blocksCount, int sentinelIndex)
{
	PaddLastBlock(bodyArray, blocksCount, sentinelIndex);
}

template <typename Predicate>
__global__ void CudaMergeAabb(Predicate ReducedAabb, ndGpuInfo* const info, cuBoundingBox* bBox, ndInt32 count)
{
	ReducedAabb(*info, bBox, count);
}

template <typename Predicate>
__global__ void CudaInitBodyArray(Predicate UpdateBodyScene, cuBodyProxy* bodyArray, cuBoundingBox* bBox)
{
	UpdateBodyScene(bodyArray, bBox);
}

template <typename Predicate>
__global__ void CudaGetBodyTransforms(Predicate GetTransform, const cuBodyProxy* const srcBuffer, cuSpatialVector* const dstBuffer, int size)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index < size)
	{
		GetTransform(srcBuffer[index], dstBuffer[index]);
	}
}

template <typename Predicate>
__global__ void CudaCountAabb(Predicate CountAabb, ndGpuInfo* const info, cuBodyProxy* bodyArray, int* scan)
{
	CountAabb(*info, bodyArray, scan);
}

template <typename Predicate>
__global__ void CudaPrefixScanSum0(Predicate PrefixScan, int* scan)
{
	PrefixScan(scan);
}

template <typename Predicate>
__global__ void CudaPrefixScanSum1(Predicate PrefixScan, int* scan, int size)
{
	PrefixScan(scan, size);
}


ndWorldSceneCuda::ndWorldSceneCuda(const ndWorldScene& src)
	:ndWorldScene(src)
	,m_context(ndCudaContext::CreateContext())
{
	m_bodyListChanged = 1;
}

ndWorldSceneCuda::~ndWorldSceneCuda()
{
	if (m_context)
	{
		delete m_context;
	}
}

void ndWorldSceneCuda::Sync()
{
	ndScene::Sync();

	//syncronize all streams before starting a new frame.
	//this is pretty horrendous function, need to find a beter method
	//hipDeviceSynchronize();
	//hipStreamSynchronize(m_context->m_stream0);
	m_context->SwapBuffers();
}

void ndWorldSceneCuda::FindCollidingPairs(ndBodyKinematic* const body)
{
	dAssert(0);
}

void ndWorldSceneCuda::FindCollidingPairs()
{
	//ndWorldScene::FindCollidingPairs();
}

void ndWorldSceneCuda::CalculateContacts(ndInt32 threadIndex, ndContact* const contact)
{
	dAssert(0);
}

void ndWorldSceneCuda::CalculateContacts()
{
	//ndWorldScene::CalculateContacts();
}

void ndWorldSceneCuda::LoadBodyData()
{
	auto UploadBodies = ndMakeObject::ndFunction([this](ndInt32 threadIndex, ndInt32 threadCount)
	{
		D_TRACKTIME();
		const ndVector minBox(ndFloat32(1.0e15f));
		const ndVector maxBox(ndFloat32(-1.0e15f));

		ndArray<cuBodyProxy>& data = m_context->m_bodyBufferCpu;
		cuHostBuffer<cuSpatialVector>& transformBufferCpu0 = m_context->m_transformBufferCpu0;
		cuHostBuffer<cuSpatialVector>& transformBufferCpu1 = m_context->m_transformBufferCpu1;

		ndArray<ndBodyKinematic*>& bodyArray = GetActiveBodyArray();
		//const ndStartEnd startEnd(bodyArray.GetCount() - 1, threadIndex, threadCount);
		const ndStartEnd startEnd(bodyArray.GetCount(), threadIndex, threadCount);
		for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
		{
			cuSpatialVector transform;
			ndBodyKinematic* const body = bodyArray[i];
			cuBodyProxy& proxi = data[i];

			// Get thansform and velocity
			proxi.m_mass = body->GetMassMatrix();
			proxi.m_rotation = cuQuat(body->GetRotation());
			proxi.m_posit = body->GetGlobalGetCentreOfMass();
			proxi.m_invIntertia = body->GetInvInertia();
			proxi.m_dampCoef = body->GetCachedDamping();
			proxi.m_veloc = body->GetVelocity();
			proxi.m_omega = body->GetOmega();

			// Get scene manager data
			const ndShapeInstance& collision = body->GetCollisionShape();
			const ndShape* const shape = collision.GetShape();

			proxi.m_minAabb = minBox;
			proxi.m_maxAabb = maxBox;
			proxi.m_obbSize = shape->GetObbSize();
			proxi.m_obbOrigin = shape->GetObbOrigin();
			proxi.m_scale = collision.GetScale();
			proxi.m_localPosition = collision.GetLocalMatrix().m_posit;
			proxi.m_localRotation = cuQuat(ndQuaternion(collision.GetLocalMatrix()));
			proxi.m_alignRotation = cuQuat(ndQuaternion(collision.GetAlignmentMatrix()));

			transform.m_angular = cuQuat(body->GetRotation());
			transform.m_linear = body->GetGlobalGetCentreOfMass();
			transformBufferCpu0[i] = transform;
			transformBufferCpu1[i] = transform;
		}
	});

	ndArray<cuBodyProxy>& bodyBufferCpu = m_context->m_bodyBufferCpu;
	cuDeviceBuffer<cuBodyProxy>& bodyBufferGpu = m_context->m_bodyBufferGpu;
	const ndArray<ndBodyKinematic*>& bodyArray = GetActiveBodyArray();
	cuDeviceBuffer<int>& scan = m_context->m_scan;
	cuDeviceBuffer<cuBoundingBox>& boundingBoxGpu = m_context->m_boundingBoxGpu;
	cuDeviceBuffer<cuSpatialVector>& transformBufferGpu = m_context->m_transformBufferGpu;
	cuHostBuffer<cuSpatialVector>& transformBufferCpu0 = m_context->m_transformBufferCpu0;
	cuHostBuffer<cuSpatialVector>& transformBufferCpu1 = m_context->m_transformBufferCpu1;

	const ndInt32 cpuBodyCount = bodyArray.GetCount();
	const ndInt32 gpuBodyCount = D_THREADS_PER_BLOCK * ((cpuBodyCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK);
	
	bodyBufferCpu.SetCount(cpuBodyCount);
	bodyBufferGpu.SetCount(gpuBodyCount);
	transformBufferGpu.SetCount(cpuBodyCount);
	transformBufferCpu0.SetCount(cpuBodyCount);
	transformBufferCpu1.SetCount(cpuBodyCount);
	scan.SetCount(gpuBodyCount);
	boundingBoxGpu.SetCount(gpuBodyCount / D_THREADS_PER_BLOCK);

	ParallelExecute(UploadBodies);
	bodyBufferGpu.ReadData(&bodyBufferCpu[0], cpuBodyCount);
}

void ndWorldSceneCuda::GetBodyTransforms()
{
	D_TRACKTIME();

	auto GetTransform = [] __device__(const cuBodyProxy& body, cuSpatialVector& transform)
	{
		transform.m_linear = body.m_posit;
		transform.m_angular = body.m_rotation;
	};

	if (m_context->m_bodyBufferGpu.GetCount())
	{
		cuHostBuffer<cuSpatialVector>& cpuBuffer = m_context->m_transformBufferCpu0;
		cuDeviceBuffer<cuSpatialVector>& gpuBuffer = m_context->m_transformBufferGpu;

		ndInt32 threads = m_context->m_bodyBufferGpu.GetCount();
		ndInt32 blocks = (threads + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
		cuBodyProxy* const bodiesGpu = &m_context->m_bodyBufferGpu[0];

		gpuBuffer.SetCount(threads);
		cpuBuffer.SetCount(threads);
		hipStream_t stream = m_context->m_stream0;
		CudaGetBodyTransforms <<<blocks, D_THREADS_PER_BLOCK, 0, stream>>> (GetTransform, bodiesGpu, &gpuBuffer[0], threads);
		gpuBuffer.WriteData(&cpuBuffer[0], cpuBuffer.GetCount(), stream);
	}
}

void ndWorldSceneCuda::UpdateTransform()
{
	D_TRACKTIME();
	GetBodyTransforms();

	auto SetTransform = ndMakeObject::ndFunction([this](ndInt32 threadIndex, ndInt32 threadCount)
	{
		D_TRACKTIME();
		const ndArray<ndBodyKinematic*>& bodyArray = GetActiveBodyArray();
		const cuSpatialVector* const data = &m_context->m_transformBufferCpu1[0];
		const ndStartEnd startEnd(bodyArray.GetCount() - 1, threadIndex, threadCount);
		for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
		{
			ndBodyKinematic* const body = bodyArray[i];
			const cuSpatialVector& transform = data[i];
			const ndVector position(transform.m_linear.x, transform.m_linear.y, transform.m_linear.z, ndFloat32(1.0f));
			const ndQuaternion rotation(ndVector(transform.m_angular.x, transform.m_angular.y, transform.m_angular.z, transform.m_angular.w));
			body->SetMatrixAndCentreOfMass(rotation, position);

			body->m_transformIsDirty = true;
			UpdateTransformNotify(threadIndex, body);
		}
	});
	ParallelExecute(SetTransform);
	
	//ndScene::UpdateTransform();
}

void ndWorldSceneCuda::UpdateBodyList()
{
	D_TRACKTIME();
	bool bodyListChanged = m_bodyListChanged;
	ndWorldScene::UpdateBodyList();
	if (bodyListChanged)
	{
		LoadBodyData();
		hipDeviceSynchronize();
	}
}

void ndWorldSceneCuda::InitBodyArray()
{
//	ndWorldScene::InitBodyArray();

	D_TRACKTIME();
	// this has to be recreated in gpu
	//ndInt32 scans[D_MAX_THREADS_COUNT][2];
	//auto BuildBodyArray = ndMakeObject::ndFunction([this, &scans](ndInt32 threadIndex, ndInt32 threadCount)
	//{
	//	D_TRACKTIME();
	//	const ndArray<ndBodyKinematic*>& view = GetActiveBodyArray();
	//
	//	ndInt32* const scan = &scans[threadIndex][0];
	//	scan[0] = 0;
	//	scan[1] = 0;
	//
	//	const ndFloat32 timestep = m_timestep;
	//	const ndStartEnd startEnd(view.GetCount() - 1, threadIndex, threadCount);
	//	for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
	//	{
	//		ndBodyKinematic* const body = view[i];
	//		body->ApplyExternalForces(threadIndex, timestep);
	//
	//		body->PrepareStep(i);
	//		UpdateAabb(threadIndex, body);
	//
	//		const ndInt32 key = body->m_sceneEquilibrium;
	//		scan[key] ++;
	//	}
	//});
	
	//auto CompactMovingBodies = ndMakeObject::ndFunction([this, &scans](ndInt32 threadIndex, ndInt32 threadCount)
	//{
	//	D_TRACKTIME();
	//	const ndArray<ndBodyKinematic*>& activeBodyArray = GetActiveBodyArray();
	//	ndBodyKinematic** const sceneBodyArray = &m_sceneBodyArray[0];
	//
	//	const ndArray<ndBodyKinematic*>& view = m_bodyList.m_view;
	//	ndInt32* const scan = &scans[threadIndex][0];
	//
	//	const ndStartEnd startEnd(view.GetCount(), threadIndex, threadCount);
	//	for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
	//	{
	//		ndBodyKinematic* const body = activeBodyArray[i];
	//		const ndInt32 key = body->m_sceneEquilibrium;
	//		const ndInt32 index = scan[key];
	//		sceneBodyArray[index] = body;
	//		scan[key] ++;
	//	}
	//});
	
	//ParallelExecute(BuildBodyArray);
	//ndInt32 sum = 0;
	//ndInt32 threadCount = GetThreadCount();
	//for (ndInt32 j = 0; j < 2; j++)
	//{
	//	for (ndInt32 i = 0; i < threadCount; ++i)
	//	{
	//		const ndInt32 count = scans[i][j];
	//		scans[i][j] = sum;
	//		sum += count;
	//	}
	//}
	//
	//ndInt32 movingBodyCount = scans[0][1] - scans[0][0];
	//m_sceneBodyArray.SetCount(m_bodyList.GetCount());
	//if (movingBodyCount)
	//{
	//	ParallelExecute(CompactMovingBodies);
	//}
	//
	//m_sceneBodyArray.SetCount(movingBodyCount);
	//
	//ndBodyKinematic* const sentinelBody = m_sentinelBody;
	//sentinelBody->PrepareStep(GetActiveBodyArray().GetCount() - 1);
	//
	//sentinelBody->m_isStatic = 1;
	//sentinelBody->m_autoSleep = 1;
	//sentinelBody->m_equilibrium = 1;
	//sentinelBody->m_equilibrium0 = 1;
	//sentinelBody->m_isJointFence0 = 1;
	//sentinelBody->m_isJointFence1 = 1;
	//sentinelBody->m_isConstrained = 0;
	//sentinelBody->m_sceneEquilibrium = 1;
	//sentinelBody->m_weigh = ndFloat32(0.0f);

	auto ReducedAabb = [] __device__(ndGpuInfo& info, cuBoundingBox* bBoxOut, int index)
	{
		__shared__  cuBoundingBox aabb[D_THREADS_PER_BLOCK];

		aabb[threadIdx.x].m_min = bBoxOut[threadIdx.x].m_min;
		aabb[threadIdx.x].m_max = bBoxOut[threadIdx.x].m_max;
		__syncthreads();

		if (threadIdx.x >= index)
		{
			aabb[threadIdx.x] = aabb[index - 1];
		}
		__syncthreads();

		for (int i = D_THREADS_PER_BLOCK / 2; i; i = i >> 1)
		{
			if (threadIdx.x < i)
			{
				aabb[threadIdx.x].m_min = aabb[threadIdx.x].m_min.Min(aabb[threadIdx.x + i].m_min);
				aabb[threadIdx.x].m_max = aabb[threadIdx.x].m_max.Max(aabb[threadIdx.x + i].m_max);
			}
			__syncthreads();
		}

		if (threadIdx.x == 0)
		{
			cuVector minBox((aabb[0].m_min.Scale(D_CUDA_SCENE_INV_GRID_SIZE).Floor()).Scale(D_CUDA_SCENE_GRID_SIZE));
			cuVector maxBox((aabb[0].m_max.Scale(D_CUDA_SCENE_INV_GRID_SIZE).Floor()).Scale(D_CUDA_SCENE_GRID_SIZE) + cuVector(D_CUDA_SCENE_GRID_SIZE));
			minBox.w = 0.0f;
			maxBox.w = 0.0f;
			info.m_worldBox.m_min = minBox;
			info.m_worldBox.m_max = maxBox;
		}
	};

	auto PaddLastBodyBlock = [] __device__(cuBodyProxy* bodyArray, int blocksCount, int sentinelIndex)
	{
		int index = (blocksCount - 1) * blockDim.x + threadIdx.x;
		if (index == sentinelIndex)
		{
			bodyArray[sentinelIndex].m_posit = bodyArray[sentinelIndex - 1].m_posit;
			bodyArray[sentinelIndex].m_rotation = bodyArray[sentinelIndex - 1].m_rotation;
		}
		__syncthreads();

		if (index > sentinelIndex)
		{
			bodyArray[index].m_rotation = bodyArray[sentinelIndex].m_rotation;
			bodyArray[index].m_posit = bodyArray[sentinelIndex].m_posit;
			bodyArray[index].m_obbSize = bodyArray[sentinelIndex].m_obbSize;
			bodyArray[index].m_obbOrigin = bodyArray[sentinelIndex].m_obbOrigin;
			bodyArray[index].m_scale = bodyArray[sentinelIndex].m_scale;
			bodyArray[index].m_localPosition = bodyArray[sentinelIndex].m_localPosition;
			bodyArray[index].m_localRotation = bodyArray[sentinelIndex].m_localRotation;
			bodyArray[index].m_alignRotation = bodyArray[sentinelIndex].m_alignRotation;
		}
	};

	auto UpdateAabb = [] __device__(cuBodyProxy* bodyArray, cuBoundingBox* bBox)
	{
		__shared__  cuBoundingBox aabb[D_THREADS_PER_BLOCK];

		int index = threadIdx.x + blockDim.x * blockIdx.x;
		cuBodyProxy& body = bodyArray[index];

		// calculate shape global Matrix
		body.m_globalSphapeRotation = body.m_localRotation * body.m_rotation;
		cuMatrix3x3 matrix(body.m_globalSphapeRotation.GetMatrix3x3());
		body.m_globalSphapePosition = matrix.RotateVector(body.m_localPosition) + body.m_posit;

		// calculate world aabb
		//ndMatrix scaleMatrix;
		//scaleMatrix[0] = matrix[0].Scale(m_scale.m_x);
		//scaleMatrix[1] = matrix[1].Scale(m_scale.m_y);
		//scaleMatrix[2] = matrix[2].Scale(m_scale.m_z);
		//scaleMatrix[3] = matrix[3];
		//scaleMatrix = m_alignmentMatrix * scaleMatrix;
		matrix.m_front = matrix.m_front.Scale(body.m_scale.x);
		matrix.m_up    = matrix.m_up.Scale(body.m_scale.y);
		matrix.m_right = matrix.m_right.Scale(body.m_scale.z);
		matrix = body.m_alignRotation.GetMatrix3x3() * matrix;

		//const ndVector size0(m_shape->GetObbSize());
		//const ndVector size(scaleMatrix.m_front.Abs().Scale(size0.m_x) + scaleMatrix.m_up.Abs().Scale(size0.m_y) + scaleMatrix.m_right.Abs().Scale(size0.m_z));
		//const ndVector origin(scaleMatrix.TransformVector(m_shape->GetObbOrigin()));
		const cuVector origin(matrix.RotateVector(body.m_obbOrigin) + body.m_globalSphapePosition);
		const cuVector size(matrix.m_front.Abs().Scale(body.m_obbSize.x) + matrix.m_up.Abs().Scale(body.m_obbSize.y) + matrix.m_right.Abs().Scale(body.m_obbSize.z));

		//p0 = (origin - size - m_padding) & ndVector::m_triplexMask;
		//p1 = (origin + size + m_padding) & ndVector::m_triplexMask;
		const cuVector padding(1.0f / 16.0f);
		const cuVector minBox(origin - size - padding);
		const cuVector maxBox(origin + size + padding);


		int threadId = threadIdx.x;
		// save aabb and calculate bonding box for this thread block
		body.m_minAabb = minBox;
		body.m_maxAabb = maxBox;
		aabb[threadId].m_min = minBox;
		aabb[threadId].m_max = maxBox;
		__syncthreads();

		for (int i = D_THREADS_PER_BLOCK / 2; i; i = i >> 1)
		{
			if (threadId < i)
			{
				aabb[threadId].m_min = aabb[threadIdx.x].m_min.Min(aabb[threadId + i].m_min);
				aabb[threadId].m_max = aabb[threadIdx.x].m_max.Max(aabb[threadId + i].m_max);
			}
			__syncthreads();
		}

		if (threadId == 0)
		{
			bBox[blockIdx.x].m_min = aabb[0].m_min;
			bBox[blockIdx.x].m_max = aabb[0].m_max;
		}
	};

	auto CountAabb = [] __device__(const ndGpuInfo& info, const cuBodyProxy* bodyArray, int* scan)
	{
		__shared__  cuBoundingBox cacheAabb;
		if (threadIdx.x == 0)
		{
			cacheAabb.m_min = info.m_worldBox.m_min;
			cacheAabb.m_max = info.m_worldBox.m_max;
		}
		__syncthreads();

		const cuVector minBox(cacheAabb.m_min);
		//const cuVector maxBox(cacheAabb.m_max);

		int index = threadIdx.x + blockDim.x * blockIdx.x;
		const cuVector bodyBoxMin(bodyArray[index].m_minAabb);
		const cuVector bodyBoxMax(bodyArray[index].m_maxAabb);

		int x0 = __float2int_rd((bodyBoxMin.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE);
		int y0 = __float2int_rd((bodyBoxMin.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE);
		int z0 = __float2int_rd((bodyBoxMin.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE);
		int x1 = __float2int_rd((bodyBoxMax.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
		int y1 = __float2int_rd((bodyBoxMax.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
		int z1 = __float2int_rd((bodyBoxMax.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
		int count = (z1 - z0) * (y1 - y0) * (x1 - x0);
		scan[index] = count;
	};

	auto PrefixScanSum0 = [] __device__(int* scan)
	{
		__shared__  int cacheBuffer[2 * D_THREADS_PER_BLOCK];

		int index = threadIdx.x + blockDim.x * blockIdx.x;

		cacheBuffer[threadIdx.x] = 0;
		int threadId = threadIdx.x + D_THREADS_PER_BLOCK;
		cacheBuffer[threadId] = scan[index];
		__syncthreads();
		
		for (int i = 1; i < D_THREADS_PER_BLOCK; i = i << 1)
		{
			int sum = cacheBuffer[threadId] + cacheBuffer[threadId - i];
			__syncthreads();
			cacheBuffer[threadId] = sum;
			__syncthreads();
		}
		scan[index] = cacheBuffer[threadId];
	};

	auto PrefixScanSum1 = [] __device__(int* scan, int size)
	{
		int threadId = threadIdx.x;
		const int blocks = size / D_THREADS_PER_BLOCK;
		for (int i = 1; i < blocks; i ++)
		{
			int sum = scan[i * D_THREADS_PER_BLOCK - 1];
			__syncthreads();
			scan[i * D_THREADS_PER_BLOCK + threadId] += sum;
			__syncthreads();
		}
	};


	ndGpuInfo* const info = m_context->m_sceneInfo;
	hipStream_t stream = m_context->m_stream0;
	ndInt32 threads = m_context->m_bodyBufferGpu.GetCount();
	ndInt32 blocksCount = (threads + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	dAssert(blocksCount < D_THREADS_PER_BLOCK);
	dAssert(blocksCount * D_THREADS_PER_BLOCK == threads);

	int* const scan = &m_context->m_scan[0];
	cuBodyProxy* const bodiesGpu = &m_context->m_bodyBufferGpu[0];
	cuBoundingBox* const bBoxGpu = &m_context->m_boundingBoxGpu[0];

	ndInt32 sentinelIndex = m_context->m_bodyBufferCpu.GetCount() - 1;
	CudaAddBodyPadding << <1, D_THREADS_PER_BLOCK, 0, stream >> > (PaddLastBodyBlock, bodiesGpu, blocksCount, sentinelIndex);
	CudaInitBodyArray << <blocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (UpdateAabb, bodiesGpu, bBoxGpu);
	CudaMergeAabb << <1, D_THREADS_PER_BLOCK, 0, stream >> > (ReducedAabb, info, bBoxGpu, blocksCount);

	CudaCountAabb << <blocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (CountAabb, info, bodiesGpu, scan);
	CudaPrefixScanSum0 << <blocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (PrefixScanSum0, scan);
	CudaPrefixScanSum1 << <1, D_THREADS_PER_BLOCK, 0, stream >> > (PrefixScanSum1, scan, threads);
}
